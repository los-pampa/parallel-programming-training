#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 4096
#define M 4096
#define K 4096

void cpu_matrix_multiplication(float *A, float *B, float *C){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < M; j++){
            for(int k = 0; k<K; k++){
                C[(i*N)+j] += A[(i*M)+k] * B[(k*N)+j];
            }
        }
    }
}

void print_matrix(float *C) {
    for(int i=0; i< N; i++) {
        for(int j=0; j<M; j++){
            printf("%f ", C[(i*N)+j]);
        }
        printf("\n");
    }
}


__global__ void gpu_matrix_multiplication(int *A, int *B, int *C){
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    for(int k = 0; k<K; k++){
        C[(linha*N)+coluna] += A[(linha*M)+k] * B[(k*N)+coluna];
    }

}

int main(int argc, char **argv) {
    float *A, *B, *C;
    float *A_d, *B_d, *C_d;
    /* Alocação na CPU */
    A = (float*) malloc(sizeof(float)*M*K);
    B = (float*) malloc(sizeof(float)*K*N);
    C = (float*) malloc(sizeof(float)*N*M);

    /* Alocação na GPU */
    hipMalloc(&A_d, M*K*sizeof(float));
    hipMalloc(&B_d, K*N*sizeof(float));
    hipMalloc(&C_d, M*N*sizeof(float));

    /*preenche as matrizes com valores iniciais */
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            A[(i*N)+j] = (float) i;
            B[(i*N)+j] = (float) j;
            C[(i*N)+j] = 0.0;
        }
    }


    /* Copiar dos da CPU para a GPU */
    hipMemcpy(A_d, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, K*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, M*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32,32)
    dim3 blocksPerGrid(N/threadsPerBlock.x, N/threadsPerBlock.y)

    gpu_matrix_multiplication<<<blocksPerGrid, threadsPerBlock>>>();
    //cpu_matrix_multiplication();

    hipMemcpy(C, C_d, M*N*sizeof(float), hipMemcpyDeviceToHost);

    print_matrix(C);
    
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    
    free(A);
    free(B);
    free(C);
}

















































    gpu_matrix_multiplicacao<gridXblocoYY>>>(A, B, C);    // 

    cpu_matrix_multiplication(A, B, C);

    print_matrix(C);
    return 0;
}